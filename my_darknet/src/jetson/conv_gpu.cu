#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#include "hipblas.h"

#include "type.h"

void checkCUDA(hipError_t error)
{
  if (error != (hipError_t)hipSuccess) {
    std::cerr << "[ERROR] CUDA " << error << std::endl;
    exit(0);
  }
}

void checkCuBLAS(hipblasStatus_t status)
{
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "[ERROR] cuBLAS ";
    switch (status) {
      case HIPBLAS_STATUS_NOT_INITIALIZED:  std::cerr << "not initialized";  break;
      case HIPBLAS_STATUS_ALLOC_FAILED:     std::cerr << "alloc failed";     break;
      case HIPBLAS_STATUS_INVALID_VALUE:    std::cerr << "invalid value";    break;
      case HIPBLAS_STATUS_ARCH_MISMATCH:    std::cerr << "arch mismatch";    break;
      case HIPBLAS_STATUS_MAPPING_ERROR:    std::cerr << "mapping error";    break;
      case HIPBLAS_STATUS_EXECUTION_FAILED: std::cerr << "execution failed"; break;
      case HIPBLAS_STATUS_INTERNAL_ERROR:   std::cerr << "internal error";   break;
      case HIPBLAS_STATUS_NOT_SUPPORTED:    std::cerr << "not supported";    break;
      case HIPBLAS_STATUS_UNKNOWN:    std::cerr << "license error";    break;
      default:                             std::cerr << "unknown error";    break;
    }
    std::cerr << std::endl;
    exit(0);
  }
}

#ifdef CUBLAS
extern "C" void cublas_sgemm(int M, int N, int K, scalar_t* A, int lda, scalar_t* B, int ldb, scalar_t* C, int ldc)
{
  hipblasHandle_t handle;
  scalar_t *A_d, *B_d, *C_d;

  checkCuBLAS(hipblasCreate(&handle));

  checkCUDA(hipMalloc((void**)&A_d, sizeof(scalar_t)*N*K));
  checkCUDA(hipMalloc((void**)&B_d, sizeof(scalar_t)*K*M));
  checkCUDA(hipMalloc((void**)&C_d, sizeof(scalar_t)*N*M));

  const float alpha = 1.0f;
  const float beta = 1.0f;

  checkCuBLAS(hipblasSetMatrix(N, K, sizeof(scalar_t), B, ldb, A_d, ldb));
  checkCuBLAS(hipblasSetMatrix(K, M, sizeof(scalar_t), A, lda, B_d, lda));
  checkCuBLAS(hipblasSetMatrix(N, M, sizeof(scalar_t), C, ldc, C_d, ldc));

  checkCuBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, A_d, ldb, B_d, lda, &beta, C_d, ldc));

  checkCuBLAS(hipblasGetMatrix(N, M, sizeof(scalar_t), C_d, ldc, C, ldc));

  checkCUDA(hipFree(A_d));
  checkCUDA(hipFree(B_d));
  checkCUDA(hipFree(C_d));

  hipblasDestroy(handle);
}

void cublas_sgemm_except_memcpy(int M, int N, int K, scalar_t* A, int lda, scalar_t* B, int ldb, scalar_t* C, int ldc)
{
  hipblasHandle_t handle;
  checkCuBLAS(hipblasCreate(&handle));
  const float alpha = 1.0f;
  const float beta = 1.0f;

  checkCuBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, ldb, A, lda, &beta, C, ldc));

  hipblasDestroy(handle);
}
#endif

__global__ void conv2d_cuda(scalar_t* INPUT, scalar_t* WEIGHT, scalar_t* OUTPUT,
                            int out_ch, int in_ch, int k_size, int in_h, int in_w,
                            int stride, int l_pad, int r_pad, int u_pad, int d_pad)
{
  int out_w = (in_w + l_pad+r_pad - k_size) / stride + 1;
  int out_h = (in_h + u_pad+d_pad - k_size) / stride + 1;

  if (blockIdx.x < out_w*out_h) { // N
    if (blockIdx.y < out_ch) { // M
      if (threadIdx.x < in_ch*k_size*k_size) { // K
        int w_offset = threadIdx.x % k_size;
        int h_offset = (threadIdx.x / k_size) % k_size;
        int c_offset = threadIdx.x / k_size / k_size;
        int ot_w_idx = blockIdx.x % out_w;
        int ot_h_idx = blockIdx.x / out_w;
        int w = ot_w_idx*stride + w_offset - l_pad;
        int h = ot_h_idx*stride + h_offset - u_pad;
        int in_idx = w + in_w*(h + in_h*c_offset);
        int wt_idx = threadIdx.x + in_ch*k_size*k_size*blockIdx.y;
        int ot_idx = blockIdx.x + out_w*out_h*blockIdx.y;
        if (w >= 0 && w < in_w && h >= 0 && h < in_h) {
          OUTPUT[ot_idx] += INPUT[in_idx] * WEIGHT[wt_idx];
        }
      }
    }
  }
  /*
  if (threadIdx.x < out_ch) { // M
    if (blockIdx.y < in_ch*k_size*k_size) { // K
      if (blockIdx.x < out_w*out_h) { // N
        int w_offset = blockIdx.y % k_size;
        int h_offset = (blockIdx.y / k_size) % k_size;
        int c_offset = blockIdx.y / k_size / k_size;
        int ot_w_idx = blockIdx.x % out_w;  // output w index
        int ot_h_idx = blockIdx.x / out_w;  // output h index
        int w = ot_w_idx*stride + w_offset - l_pad;  // input w index
        int h = ot_h_idx*stride + h_offset - u_pad;  // input h index
        int in_idx = w + in_w*(h + in_h*c_offset);
        int wt_idx = blockIdx.y + in_ch*k_size*k_size*threadIdx.x;
        int ot_idx = blockIdx.x + out_w*out_h*threadIdx.x;
        if (w >= 0 && w < in_w && h >= 0 && h < in_h) {
          OUTPUT[ot_idx] += INPUT[in_idx] * WEIGHT[wt_idx];
        }
      }
    }
  }
*/
}

__global__ void im2col_gpu(scalar_t* data_im, scalar_t* data_col, int channel, int height, int width, int ksize,
                           int stride, int l_pad, int r_pad, int u_pad, int d_pad)
{
  /*
  int height_col = (height + u_pad+d_pad - ksize) / stride + 1;
  int width_col  = (width  + l_pad+r_pad - ksize) / stride + 1;

  if (blockIdx.y < channel * ksize * ksize) {
    if (blockIdx.x < width_col * height_col) {
      int w_offset = blockIdx.y % ksize;
      int h_offset = (blockIdx.y / ksize) % ksize;
      int c_im = blockIdx.y / ksize / ksize;
      int w = blockIdx.x % width_col;
      int h = blockIdx.x / width_col;
      int im_row = h_offset + h*stride - u_pad;
      int im_col = w_offset + w*stride - l_pad;
      if (im_row >= 0 && im_row < height && im_col >= 0 && im_col < width) {
        data_col[blockIdx.x + width_col*height_col*blockIdx.y] = data_im[im_col + width*(im_row + height*c_im)];
      } else {
        data_col[blockIdx.x + width_col*height_col*blockIdx.y] = 0;
      }

    }
  }
  */
  int k_w, k_h;
  for (k_h = 0; k_h < ksize; ++k_h){
    for (k_w = 0; k_w < ksize; ++k_w){
      int col_index = gridDim.x*gridDim.y*ksize*ksize*threadIdx.x + gridDim.x*gridDim.y*(ksize*k_h+k_w)
        + gridDim.x*blockIdx.y + blockIdx.x;
      int im_row = k_h + blockIdx.y*stride;
      int im_col = k_w + blockIdx.x*stride;
      int row = im_row - u_pad;
      int col = im_col - l_pad;
      if (row < 0 || col < 0 || row >= height || col >= width) {
        data_col[col_index]=0;
      }
      else {
        data_col[col_index] = data_im[col+width*(row+height*threadIdx.x)];
      }
    }
  }
}

extern "C" void conv2d_gpu(scalar_t* INPUT, scalar_t* WEIGHT, scalar_t* OUTPUT,
                           int out_ch, int in_ch, int k_size, int in_h, int in_w,
                           int stride, int l_pad, int r_pad, int u_pad, int d_pad)
{
  int out_w = (in_w + l_pad+r_pad - k_size) / stride + 1;
  int out_h = (in_h + u_pad+d_pad - k_size) / stride + 1;

#ifdef CUDA
  conv2d_cuda<<<dim3(out_w*out_h, in_ch*k_size*k_size), out_ch>>>(INPUT, WEIGHT, OUTPUT,
                                                                  out_ch, in_ch, k_size, in_h, in_w,
                                                                  stride, l_pad, r_pad, u_pad, d_pad);
#elif defined(CUBLAS)
  scalar_t* data_col;
  checkCUDA(hipMalloc((void**)&data_col, sizeof(scalar_t)*out_w*out_h*in_ch*k_size*k_size));
  //im2col_gpu<<<dim3(out_w*out_h, in_ch*k_size*k_size), 1>>>(INPUT, data_col, in_ch, in_h, in_w, k_size, stride, l_pad, r_pad, u_pad, d_pad);
  im2col_gpu<<<dim3(out_w, out_h), in_ch>>>(INPUT, data_col, in_ch, in_h, in_w, k_size, stride, l_pad, r_pad, u_pad, d_pad);

  int M = out_ch, N = out_h*out_w, K = in_ch*k_size*k_size;
  cublas_sgemm_except_memcpy(M, N, K, WEIGHT, K, data_col, N, OUTPUT, N);

  checkCUDA(hipFree(data_col));
#endif

}

extern "C" void conv2d_gpu_except_memcpy( scalar_t* INPUT, scalar_t* WEIGHT, scalar_t* OUTPUT,
                                          int out_ch, int in_ch, int k_size, int in_h, int in_w,
                                          int stride, int l_pad, int r_pad, int u_pad, int d_pad)
{
  int out_w = (in_w + l_pad+r_pad - k_size) / stride + 1;
  int out_h = (in_h + u_pad+d_pad - k_size) / stride + 1;

  conv2d_cuda<<<dim3(out_w*out_h, in_ch*k_size*k_size), out_ch>>>(INPUT, WEIGHT, OUTPUT,
                                                                  out_ch, in_ch, k_size, in_h, in_w,
                                                                  stride, l_pad, r_pad, u_pad, d_pad);
}
