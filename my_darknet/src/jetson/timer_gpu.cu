#include <hip/hip_runtime.h>

#include <stdio.h>

extern void checkCUDA(hipError_t error);

static hipEvent_t start, stop;
static hipEvent_t loc_start, loc_stop;

extern "C" void init_timer(void)
{
  checkCUDA(hipEventCreate(&start));
  checkCUDA(hipEventCreate(&stop));
}

extern "C" void init_local_timer(void)
{
  checkCUDA(hipEventCreate(&loc_start));
  checkCUDA(hipEventCreate(&loc_stop));
}

extern "C" void start_timer(void)
{
  checkCUDA(hipEventRecord(start, 0));
}

extern "C" void stop_timer(float* ms)
{
  checkCUDA(hipEventRecord(stop, 0));
  checkCUDA(hipEventSynchronize(stop));
  checkCUDA(hipEventElapsedTime(ms, start, stop));
}

extern "C" void start_local_timer(void)
{
  checkCUDA(hipEventRecord(loc_start, 0));
}

extern "C" void stop_local_timer(float* ms)
{
  checkCUDA(hipEventRecord(loc_stop, 0));
  checkCUDA(hipEventSynchronize(loc_stop));
  checkCUDA(hipEventElapsedTime(ms, loc_start, loc_stop));
}

extern "C" void free_timer(void)
{
  checkCUDA(hipEventDestroy(start));
  checkCUDA(hipEventDestroy(stop));
}

extern "C" void free_local_timer(void)
{
  checkCUDA(hipEventDestroy(loc_start));
  checkCUDA(hipEventDestroy(loc_stop));
}
